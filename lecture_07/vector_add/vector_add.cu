#include <hip/hip_runtime.h>
#include <random>
#include <vector>
#include <iostream>
#include <exception>

/**
 * TODO: write a kernel that does the vector addition C = A + B with 1 thread
 */
__global__ void vectorAddOneThread(const float *A, const float *B, float *C, int N) {
  for(int i = 0; i < N; i++) {
    C[i] = A[i] + B[i];
  }
}

/**
 * TODO: write a kernel that does the vector addition C = A + B with 1 Block
 * and 256 threads Hint: When 256 threads are working on one loop how the loop
 * changes?
 */
__global__ void vectorAddOneBlock(const float *A, const float *B, float *C, int N) {
  for(int i = 0; i < N / blockDim.x; i++) {
    int idx = i * blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
  }
}

/**
 * TODO: write a kernel that does the vector addition C = A+B with grid of
 * blocks. Each block has 256 threads. Hint: what check do you need to implement
 * to avoid invalid memory reference?
 */
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    C[idx] = A[idx] + B[idx];
  }
}

/* -------------------------------------------------------------------------- */
void checkResults(std::string test, const float *A, const float *B,
                  const float *C, int N) {
  // Verify that the result vector is correct
  for (int i = 0; i < N; ++i) {
    if (std::abs(A[i] + B[i] - C[i]) > 1e-5) {
      throw std::runtime_error("Result verification failed at element "
                               + std::to_string(i) + " for test " + test);
    }
  }
}

/**
 * Host main routine
 */
int main() {
  // Print the vector length to be used, and compute its size
  int N = 1 << 20; // 1M elements
  size_t size_in_bytes = N * sizeof(float);
  std::cout << "[Vector addition of " << N << " elements]" << std::endl;

  float *d_A{nullptr};
  float *d_B{nullptr};
  float *d_C{nullptr};

  // TODO: allocate d_A, d_B, and d_C
  hipMallocManaged(&d_A, size_in_bytes);
  hipMallocManaged(&d_B, size_in_bytes);
  hipMallocManaged(&d_C, size_in_bytes);

  std::mt19937 gen(2006);
  std::uniform_real_distribution<> dis(0.f, 1.f);

  // Initialize the input vectors
  for (int i = 0; i < N; ++i) {
    d_A[i] = dis(gen);
    d_B[i] = dis(gen);
  }

  // Launch the Vector Add CUDA Kernel
  int threads_per_block = 256;

  // TODO: Launch the Vector Add CUDA Kernel with one threads
  vectorAddOneThread <<<1, 1>>> (d_A, d_B, d_C, N);
  hipDeviceSynchronize(); // Since kernel launches is async wrt to the host we
                           // have to syncronize

  checkResults("vectorAddOneThread", d_A, d_B, d_C, N);

  // TODO: Launch the Vector Add CUDA Kernel with one block and 256 threads
  vectorAddOneBlock <<<1, threads_per_block>>> (d_A, d_B, d_C, N);
  hipDeviceSynchronize();

  checkResults("vectorAddOneBlock", d_A, d_B, d_C, N);

  int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;; // TODO: compute the blocks per grid
  // TODO: Launch the Vector Add CUDA Kernel with blocksPerGrid and 256 threads
  vectorAdd <<<blocks_per_grid, threads_per_block>>> (d_A, d_B, d_C, N);
  hipDeviceSynchronize();

  checkResults("vectorAdd", d_A, d_B, d_C, N);

  std::cout << "Test PASSED" << std::endl;

  // TODO: Free device global memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  std::cout << "Done" << std::endl;
  return 0;
}
