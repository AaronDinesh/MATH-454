// CUDA runtime
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_world() {
  printf("Hello world (%d %d %d) - (%d %d %d)\n",
         threadIdx.x, threadIdx.y, threadIdx.z,
         blockIdx.x, blockIdx.y, blockIdx.z);
}

int main() {
  printf("printf() is called. Output:\n");

  // Kernel configuration, where a two-dimensional grid and
  // three-dimensional blocks are configured.
  dim3 dimGrid(2, 2);
  dim3 dimBlock(2, 2, 2);

  hello_world<<<dimGrid, dimBlock>>>();
  //Wait here for everything to be printed out.
  hipDeviceSynchronize();

  return 0;
}
